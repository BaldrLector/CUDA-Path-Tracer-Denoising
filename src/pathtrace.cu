#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "main.h"
#include "denoise.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int frame, int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 27) | (frame << 13) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes two images to the OpenGL PBO directly.
__global__ void sendTwoImagesToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* leftImage, glm::vec3* rightImage) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        int leftIndex = x + (y * resolution.x * 2);
        int rightIndex = x + (y * resolution.x * 2) + resolution.x;
        
        glm::vec3 pix;
        glm::ivec3 color;

        // write to left (path traced) image pixel locations.
        pix = leftImage[index];
        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);
        pbo[leftIndex].w = 0;
        pbo[leftIndex].x = color.x;
        pbo[leftIndex].y = color.y;
        pbo[leftIndex].z = color.z;

        // write to right (denoised) image pixel locations.
        pix = rightImage[index];
        color.x = glm::clamp((int)(pix.x * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z * 255.0), 0, 255);
        pbo[rightIndex].w = 0;
        pbo[rightIndex].x = color.x;
        pbo[rightIndex].y = color.y;
        pbo[rightIndex].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;

static Triangle * dev_triangles = NULL;                           // triangles
static GBufferTexel * dev_gbuffer = NULL;                         // G-buffer for normal and depth
static glm::vec3 * dev_denoised_image = NULL;

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(Triangle));
    hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);
    
    hipMalloc(&dev_gbuffer, pixelcount * sizeof(GBufferTexel));

    hipMalloc(&dev_denoised_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_denoised_image, 0, pixelcount * sizeof(glm::vec3));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);

    hipFree(dev_triangles);
    hipFree(dev_gbuffer);
    hipFree(dev_denoised_image);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);

        // initial ray
		PathSegment & segment = pathSegments[index];
        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f);
        segment.ray.direction = glm::normalize(cam.view
        - cam.right * cam.pixelLength.x * ((float)x - (float)(cam.resolution.x * 0.5f - 0.5f))
        - cam.up * cam.pixelLength.y * ((float)y - (float)(cam.resolution.y * 0.5f - 0.5f))
        );
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
	}
}

__host__ __device__
bool computeIntersection(Ray& ray, ShadeableIntersection& intersection, Geom * geoms, int geoms_size, Triangle* triangles) {
    // closest hit
    float t_min = FLT_MAX;
    int hit_geom_index = -1;
    glm::vec3 intersect_point;
    glm::vec3 normal;
    bool outside;

    float t;
    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;
    bool tmp_outside;

    for (int i = 0; i < geoms_size; i++)
    {
        Geom & geom = geoms[i];
        if (geom.type == CUBE) t = boxIntersectionTest(geom, ray, tmp_intersect, tmp_normal, tmp_outside);
        else if (geom.type == SPHERE) t = sphereIntersectionTest(geom, ray, tmp_intersect, tmp_normal, tmp_outside);
        else if (geom.type == MESH) t = meshIntersectionTest(geom, triangles, ray, tmp_intersect, tmp_normal, tmp_outside);

        // update closest hit
        if (t > 0.0f && t < t_min) {
            t_min = t;
            hit_geom_index = i;
            intersect_point = tmp_intersect;
            normal = tmp_normal;
            outside = tmp_outside;
        }
    }

    if (hit_geom_index == -1) {
        // The ray hits nothing
        intersection.t = -1.0f;
        intersection.geomId = -1;
        return false;
    } else {
        //The ray hits something
        intersection.t = t_min;
        intersection.materialId = geoms[hit_geom_index].materialid;
        intersection.surfaceNormal = normal;
        intersection.geomId = hit_geom_index;
        intersection.outside = outside;
        return true;
    }
}

// TODO
__host__ __device__
void computeShadowRay(Ray& shadowRay, glm::vec3 originPos, Geom& light, unsigned int& seed) {
    // random sample in a unit circle prependiculer to the direction to light
    glm::vec3 directionToCenter = glm::normalize(light.translation - originPos);
    glm::quat rot = glm::rotation(glm::vec3(0.0f, 0.0f, 1.0f), directionToCenter);
    float theta = 2 * PI * nextRand(seed);
    glm::vec3 sampleDirection = glm::rotate(rot, glm::vec3(cosf(theta), sinf(theta), 0.0f));

    float lightRadius = 0.5f;
    shadowRay.origin = originPos;
    shadowRay.direction = glm::normalize(light.translation + sampleDirection * lightRadius - originPos);
}

// do ray tracing kernel
__global__ void rt(int frame, int num_paths, int max_depth,
    PathSegment * pathSegments, ShadeableIntersection * intersections, 
    Geom * geoms, int geoms_size, Triangle* triangles, Material * materials, GBufferTexel * gbuffer, glm::vec3 * image)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        PathSegment& segment = pathSegments[idx];
        ShadeableIntersection& intersection = intersections[idx];
        glm::vec3 accumulatedColor(0.0f);
        for (int depth = 1; depth <= max_depth; depth++) {
            bool hit = computeIntersection(segment.ray, intersection, geoms, geoms_size, triangles);

            // g-buffer
            if (depth == 1) {
                gbuffer[idx].position = segment.ray.origin + intersection.t * segment.ray.direction;
                gbuffer[idx].normal = intersection.surfaceNormal;
                gbuffer[idx].geomId = intersection.geomId;
            }

            if (!hit) break;

            unsigned int seed = initRand(idx, frame * depth, 16);
            Material material = materials[intersection.materialId];
            if (material.emittance > 0.0f) {  // Hit light (terminate ray)
                accumulatedColor += segment.color * material.color * material.emittance;
                break;
            }
            else {                            // Hit material (scatter ray)
                glm::vec3 intersectionPos = segment.ray.origin + intersection.t * segment.ray.direction;
                glm::vec3 intersectionNormal = intersection.surfaceNormal;

                // color mask
                segment.color *= material.color;
                glm::clamp(segment.color, glm::vec3(0.0f), glm::vec3(1.0f));

                // trace shadow ray
                if (true) {
                    Ray shadowRay;
                    float pdf;
                    computeShadowRay(shadowRay, intersectionPos + 1e-4f * intersectionNormal, geoms[0], seed);
                    ShadeableIntersection shadowRayIntersection;
                    bool shadowRayHit = computeIntersection(shadowRay, shadowRayIntersection, geoms, geoms_size, triangles);
                    if (shadowRayHit) {
                        Material shadowRayMaterial = materials[shadowRayIntersection.materialId];
                        if (shadowRayMaterial.emittance > 0.0f) {
                            glm::vec3 shadowRayIntersectionPos = shadowRay.origin + shadowRay.direction * shadowRayIntersection.t;
                            float diffuse = glm::max(0.0f, glm::dot(shadowRay.direction, intersectionNormal));
                            float shadowIntensity = 1.f / (shadowRayIntersection.t * shadowRayIntersection.t);  // TODO
                            accumulatedColor += segment.color * material.color
                                                * shadowRayMaterial.emittance * shadowRayMaterial.color
                                                * shadowIntensity * diffuse;
                        }
                    }
                }

                // bounce ray
                scatterRay(segment, intersectionPos, intersectionNormal, material, seed);
            }
        }
        image[segment.pixelIndex] = glm::clamp(accumulatedColor, glm::vec3(0.0f), glm::vec3(1.0f));
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;
    dim3 blocksPerGrid1d = (pixelcount + blockSize1d - 1) / blockSize1d;

    ///////////////////////////////////////////////////////////////////////////

    // Generate camera rays
    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d >>>(cam, iter, ui_tracedepth, dev_paths);
    checkCUDAError("generate camera ray");
    
    // Do actual ray tracing
    rt<<<blocksPerGrid1d, blockSize1d>>>(frame, pixelcount, ui_tracedepth,
        dev_paths, dev_intersections,
        dev_geoms, hst_scene->geoms.size(),
        dev_triangles, dev_materials, dev_gbuffer, dev_image);
    checkCUDAError("ray tracing");

    // Run denoiser!
    if (ui_denoise_enable) {
        denoise(iter, dev_image, dev_denoised_image, dev_gbuffer);
    } else {
        hipMemcpy(dev_denoised_image, dev_image, sizeof(glm::vec3) * pixelcount, hipMemcpyDeviceToDevice);
    }

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendTwoImagesToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image, dev_denoised_image);
    checkCUDAError("send images to PBO");

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}
